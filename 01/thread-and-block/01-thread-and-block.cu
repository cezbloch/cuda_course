#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>


__global__ void printSuccessForCorrectExecutionConfiguration()
{
  if (threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  }
  else {
    printf("Failure. Update the execution configuration as necessary.\n");
  }
}

int main()
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */

  printSuccessForCorrectExecutionConfiguration << <1, 1 >> > ();
}
