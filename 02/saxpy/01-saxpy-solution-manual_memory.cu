#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <>
#include <hip/hip_runtime.h>


#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nvprof to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = tid; i < N; i += stride)
  {
	  c[i] = 2 * a[i] + b[i];
  }    
}

int main()
{
  int *a, *b, *c, *h_a, *h_b, *h_c;

  int size = N * sizeof(int); // The total number of bytes per vector

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);

  hipHostMalloc(&h_a, size);
  hipHostMalloc(&h_b, size);
  hipHostMalloc(&h_c, size);

  // Initialize memory
  for (int i = 0; i < N; ++i)
  {
    h_a[i] = 2;
    h_b[i] = 1;
    h_c[i] = 0;
  }

  hipMemcpy(a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(b, h_b, size, hipMemcpyHostToDevice);
  hipMemcpy(c, h_c, size, hipMemcpyHostToDevice);

  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  int threads_per_block = 512;
  int number_of_blocks = props.multiProcessorCount * 512;
  //int number_of_blocks = N / threads_per_block + 1;

  saxpy << < number_of_blocks, threads_per_block >> > (a, b, c);

  auto syncErr = hipGetLastError();
  auto asyncErr = hipDeviceSynchronize();

  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipMemcpy(h_c, c, size, hipMemcpyDeviceToHost);

  // Print out the first and last 5 values of c for a quality check
  for (int i = 0; i < 5; ++i)
    printf("c[%d] = %d, ", i, h_c[i]);
  printf("\n");
  for (int i = N - 5; i < N; ++i)
    printf("c[%d] = %d, ", i, h_c[i]);
  printf("\n");

  hipFree(a); hipFree(b); hipFree(c);
  hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);
}
