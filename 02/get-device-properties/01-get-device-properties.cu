
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * Assign values to these variables so that the output string below prints the
   * requested properties of the currently active GPU.
   */

  int deviceId;
  int computeCapabilityMajor;
  int computeCapabilityMinor;
  int multiProcessorCount;
  int warpSize;

  /*
   * There should be no need to modify the output string below.
   */

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
