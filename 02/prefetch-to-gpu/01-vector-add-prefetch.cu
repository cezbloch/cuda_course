#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>


/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

void initWith(float num, float *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for (int i = 0; i < N; i++)
  {
    if (vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2 << 24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int threadsPerBlock;
  int numberOfBlocks;

  /*
   * nvprof should register performance changes when execution configuration
   * is updated.
   */

  threadsPerBlock = 1;
  numberOfBlocks = 1;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto << <numberOfBlocks, threadsPerBlock >> > (c, a, b, N);

  addVectorsErr = hipGetLastError();
  if (addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
