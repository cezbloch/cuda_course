#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <>
#include <hip/hip_runtime.h>


__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  hipStream_t stream[5];       // CUDA streams are of type `hipStream_t`.

  for (int i = 0; i < 5; ++i)
  {
    hipStreamCreate(&stream[i]); // Note that a pointer must be passed to `cudaCreateStream`.
  }

  for (int i = 0; i < 5; ++i)
  {
    printNumber << <1, 1, 0, stream[i] >> > (i);
  }

  hipDeviceSynchronize();


  for (int i = 0; i < 5; ++i)
  {
    hipStreamDestroy(stream[i]); // Note that a value, not a pointer, is passed to `cudaDestroyStream`.
  }
}
