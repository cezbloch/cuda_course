#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <>
#include <hip/hip_runtime.h>


__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  for (int i = 0; i < 5; ++i)
  {
    printNumber << <1, 1 >> > (i);
  }
  hipDeviceSynchronize();
}
