#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>


__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  for (int i = 0; i < 5; ++i)
  {
    printNumber << <1, 1 >> > (i);
  }
  hipDeviceSynchronize();
}
