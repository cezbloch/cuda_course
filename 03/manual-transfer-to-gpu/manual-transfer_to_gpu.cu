#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <>
#include <hip/hip_runtime.h>


void initWith(float num, float *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addArraysInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for (int i = 0; i < N; i++)
  {
    if (array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  const int N = 2 << 24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *h_a;
  float *h_b;
  float *h_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);
  hipHostMalloc(&h_a, size);
  hipHostMalloc(&h_b, size);
  hipHostMalloc(&h_c, size);

  int threadsPerBlock;
  int numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addArraysErr;
  hipError_t asyncErr;

  initWith(3, h_a, N);
  initWith(4, h_b, N);
  initWith(0, h_c, N);

  hipMemcpy(a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(b, h_b, size, hipMemcpyHostToDevice);
  hipMemcpy(c, h_c, size, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  addArraysInto << <numberOfBlocks, threadsPerBlock >> > (c, a, b, N);

  hipMemcpy(h_c, c, size, hipMemcpyDeviceToHost);

  addArraysErr = hipGetLastError();
  if (addArraysErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addArraysErr));

  asyncErr = hipDeviceSynchronize();
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, h_c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipHostFree(h_c);
}
